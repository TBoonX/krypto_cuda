#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
//#include <math.h>

//#define DEBUG

//Variablen
#define p 3
#define q 5
#define n 15
#define e 3
#define v 3
#define z 8
#define anzahl_Texte 384
#define count_cores 384

/*
Klartext: K
Geheimtext: G
Verschluesselung: G = K^v mod n
Entschluesselung: K = G^e mod n

Index des CUDA Kerns: blockIdx.x blockIdx.y

Ein groesserer Text soll ver- und entschluesselt werden.
Dieser wird jedoch wie folgt veraendert: nur kleine Buchstaben, keine Sonderzeichen außer . und ,
Dafuer werden die einzellnen chars in Integer umgewandelt.
Somit ist eine Verarbeitung moeglich.

*/

static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void verschluessselung(long int klartexte[], long int geheimtexte[])
{
	long int i, j, multi, x;
	
	long int block_length = anzahl_Texte/count_cores;
	
	for (i = 0 ; i < block_length; i++)
	{
		multi = x  = klartexte[i+blockIdx.x*block_length];
		for (j = 1; j < v; j++)
			x *= multi;
		
		geheimtexte[i+blockIdx.x*block_length] = x % n;
		
	}
}


__global__ void entschluessselung(long int geheimtexte[], long int klartexte_pruefung[])
{
	long int i, j, multi, x;
	
	long int block_length = anzahl_Texte/count_cores;

	for (i = 0 ; i < block_length; i++)
	{
		multi = x  = geheimtexte[i+blockIdx.x*block_length];
		for (j = 1; j < e; j++)
			x *= multi;
		
		klartexte_pruefung[i+blockIdx.x*block_length] = x % n;
		
	}
}

int main(void) {
	int i;
	hipEvent_t start, stop;
	float elapsedTime;
	
	long int klartexte[anzahl_Texte];
	long int klartexte_pruefung[anzahl_Texte];
	long int geheimtexte[anzahl_Texte];
	

	//Klartetexte Array belegen
	//rand initialisieren
	srand((unsigned)time(NULL));
	for (i = 0; i < anzahl_Texte; i ++)
	{
		klartexte[i] = rand() % 15;		//Zahlen nicht  zu gross waehlen
	}
	
	printf("Die Klartexte:\n");
	for (i = 0; i < anzahl_Texte; i++)
	{
		printf("%ld, ", klartexte[i]);
	}
	printf("\n\n");

	long int *dev_klartexte, *dev_geheimtexte, *dev_klartexte_pruefung;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));



	HANDLE_ERROR(hipEventRecord(start, 0));

        HANDLE_ERROR(hipMalloc((void **)&dev_klartexte, sizeof(klartexte)));
        HANDLE_ERROR(hipMalloc((void **)&dev_geheimtexte, sizeof(geheimtexte)));
        HANDLE_ERROR(hipMalloc((void **)&dev_klartexte_pruefung, sizeof(klartexte_pruefung)));

        HANDLE_ERROR(hipMemcpy(dev_klartexte, klartexte, sizeof(klartexte), hipMemcpyHostToDevice));

	dim3 blocks(count_cores, 1);

	verschluessselung<<<blocks, 1>>>(dev_klartexte, dev_geheimtexte);

        HANDLE_ERROR(hipMemcpy(geheimtexte, dev_geheimtexte, sizeof(geheimtexte), hipMemcpyDeviceToHost));
		
	printf("Die Klartexte wurden verschluesselt.\n\nGeheimtexte:\n");
	for (i = 0; i < anzahl_Texte; i++)
	{
		printf("%ld, ", geheimtexte[i]);
	}
	printf("\n\n");
	
	entschluessselung<<<blocks, 1>>>(dev_geheimtexte, dev_klartexte_pruefung);
	
	HANDLE_ERROR(hipMemcpy(klartexte_pruefung, dev_klartexte_pruefung, sizeof(klartexte_pruefung), hipMemcpyDeviceToHost));
		

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	
	printf("Die Geheimtexte wurden entschluesselt.\n\nKlartexte:\n");
		for (i = 0; i < anzahl_Texte; i++)
		{
			printf("%ld, ", klartexte_pruefung[i]);
		}
		printf("\n\n");

	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Elapsed time: %3.1f ms\n", elapsedTime);



	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	

	return EXIT_SUCCESS;
}
