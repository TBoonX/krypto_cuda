
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

//Variablen
#define p 3
#define q 5
#define n 15
#define e 3
#define v 3
#define z 8
#define anzahl_Zeichen 2688
#define count_cores 384

/*
Klartext: K
Geheimtext: G
Verschluesselung: G = K^v mod n
Entschluesselung: K = G^e mod n

Index des CUDA Kerns: blockIdx.x blockIdx.y

Ein groesserer Text soll ver- und entschluesselt werden.
Dieser wird jedoch wie folgt veraendert: nur kleine Buchstaben, keine Sonderzeichen außer . und ,
Dafuer werden die einzellnen chars in Integer umgewandelt (<=28) und dies halbiert und auf 3 Werte verteilt.
Somit ist eine Verarbeitung moeglich.

*/

static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void verschluessselung(long int klartexte[], long int geheimtexte[])
{
	long int i, j, multi, x;
	
	long int block_length = anzahl_Zeichen/count_cores;
	
	for (i = 0 ; i < block_length; i++)
	{
		multi = x  = klartexte[i+blockIdx.x*block_length];
		for (j = 1; j < v; j++)
			x *= multi;
		
		geheimtexte[i+blockIdx.x*block_length] = x % n;
		
	}
}


__global__ void entschluessselung(long int geheimtexte[], long int klartexte_pruefung[])
{
	long int i, j, multi, x;
	
	long int block_length = anzahl_Zeichen/count_cores;

	for (i = 0 ; i < block_length; i++)
	{
		multi = x  = geheimtexte[i+blockIdx.x*block_length];
		for (j = 1; j < e; j++)
			x *= multi;
		
		klartexte_pruefung[i+blockIdx.x*block_length] = x % n;
		
	}
}

void splitt(char text[], long int numbers[])
{
	int i;
	
	//Splitte Klartext
	for (i = 0; i < anzahl_Zeichen; i++)
	{
		long int number = (int)text[i];
		int modulo, multi;
		
		//char in int beginnend mit 0
		//Sonderzeichen
		if (number == 44)		//,
			number = 27;
		else if (number == 46)		//.
			number = 28;
		else if (number == 63)		//?
			number = 29;
		else if (number == 32)		//' '
			number = 30;
		else				//a-z
			number -= 97;
		
		modulo = number % 10;
		multi = (int)(number/10);
		
		//splitt
		if (multi == 0)
		{
			numbers[i*3] = modulo;
			numbers[i*3+1] = 0;
			numbers[i*3+2] = 0;
		}
		else if (multi == 1)
		{
			numbers[i*3] = 10;
			numbers[i*3+1] = modulo;
			numbers[i*3+2] = 0;
		}
		else
		{
			numbers[i*3] = 10;
			numbers[i*3+1] = 10;
			numbers[i*3+2] = number-20;
		}
	}
}

void unsplitt(char text[], long int numbers[])
{
	int i;
	
	//Splitte Klartext
	for (i = 0; i < anzahl_Zeichen; i++)
	{
		long int number = numbers[i*3]+numbers[i*3+1]+numbers[i*3+2];
		char t;
		
		//int in char
		//Sonderzeichen
		if (number == 27)		//,
			t = ',';
		else if (number == 28)		//.
			number = t = '.';
		else if (number == 29)		//?
			t = '\n';
		else if (number == 30)		//' '
			t = ' ';
		else				//a-z
			t = (char)number+97;
			
		text[i] = t;
	}
}

int main(void) {
	int i;
	hipEvent_t start, stop;
	float elapsedTime;
	
	char klartext[anzahl_Zeichen+1];
	char klartext2[anzahl_Zeichen+1];
	long int kt_splitted[anzahl_Zeichen*3+1];
	long int kt_splitted2[anzahl_Zeichen*3+1];
	long int *dev_kt_splitted, *dev_kt_splitted2, *dev_gt_splitted;
	
	//Debug
	printf("\na: %d   z: %d   ,: %d   .: %d   ?: %d    : %d\n\n", (int)'a', (int)'z', (int)',', (int)'.', (int)'?', (int)' ');
	

	//Klartetext erzeugen
	strcpy(klartext, "hat der alte hexenmeister?sich doch einmal wegbegeben.?und nun sollen seine geister?auch nach meinem willen leben.?seine wort und werke?merkt ich und den brauch,?und mit geistesstaerke?tu ich wunder auch.?walle. walle?manche strecke,?dass, zum zwecke,?wasser fliesse?und mit reichem, vollem schwalle?zu dem bade sich ergiesse.?und nun komm, du alter besen.?nimm die schlechten lumpenhuellen ?bist schon lange knecht gewesen:?nun erfuelle meinen willen.?auf zwei beinen stehe,?oben sei ein kopf,?eile nun und gehe?mit dem wassertopf.?walle. walle?manche strecke,?dass, zum zwecke,?wasser fliesse?und mit reichem, vollem schwalle?zu dem bade sich ergiesse.?seht, er laeuft zum ufer nieder,?wahrlich. ist schon an dem flusse,?und mit blitzesschnelle wieder?ist er hier mit raschem gusse.?schon zum zweiten male.?wie das becken schwillt.?wie sich jede schale?voll mit wasser fuellt.?stehe. stehe.?denn wir haben?deiner gaben?vollgemessen. ?ach, ich merk es. wehe. wehe.?hab ich doch das wort vergessen.?ach, das wort, worauf am ende?er das wird, was er gewesen.?ach, er laeuft und bringt behende.?waerst du doch der alte besen.?immer neue guesse?bringt er schnell herein,?ach. und hundert fluesse?stuerzen auf mich ein.?nein, nicht laenger?kann ichs lassen ?will ihn fassen.?das ist tuecke.?ach. nun wird mir immer baenger.?welche mine. welche blicke.?o du ausgeburt der hoelle.?soll das ganze haus ersaufen??seh ich ueber jede schwelle?doch schon wasserstroeme laufen.?ein verruchter besen,?der nicht hoeren will.?stock, der du gewesen,?steh doch wieder still.?willst am ende?gar nicht lassen??will dich fassen,?will dich halten?und das alte holz behende?mit dem scharfen beile spalten.?seht da kommt er schleppend wieder.?wie ich mich nur auf dich werfe,?gleich, o kobold, liegst du nieder ?krachend trifft die glatte schaerfe.?wahrlich, brav getroffen.?seht, er ist entzwei.?und nun kann ich hoffen,?und ich atme frei.?wehe. wehe.?beide teile?stehn in eile?schon als knechte?voellig fertig in die hoehe.?helft mir, ach. ihr hohen maechte.?und sie laufen. nass und naesser?wirds im saal und auf den stufen.?welch entsetzliches gewaesser.?herr und meister. hoer mich rufen.  ?ach, da kommt der meister.?herr, die not ist gross.?die ich rief, die geister?werd ich nun nicht los.?in die ecke,?besen, besen.?seids gewesen.?denn als geister?ruft euch nur zu diesem zwecke,?erst hervor der alte meister.?                                                                                                                                                                                                                                                                                                    ");

	printf("\n\nDer Klartext ist %d Zeichen lang.\n", sizeof(klartext)/sizeof(char));
	
	//Chars in ints aufsplitten
	splitt(klartext, kt_splitted);

	//Variablen der Zeitmessung erstellen
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	//Start Zeitmessung
	HANDLE_ERROR(hipEventRecord(start, 0));

	//allokieren
	HANDLE_ERROR(hipMalloc((void **)&dev_kt_splitted, sizeof(kt_splitted)));
	HANDLE_ERROR(hipMalloc((void **)&dev_kt_splitted2, sizeof(kt_splitted2)));
	HANDLE_ERROR(hipMalloc((void **)&dev_gt_splitted, sizeof(kt_splitted)));

	//kopieren
	HANDLE_ERROR(hipMemcpy(dev_kt_splitted, kt_splitted, sizeof(kt_splitted), hipMemcpyHostToDevice));

	//Block festlegen
	dim3 blocks(count_cores, 1);

	//verschluesseln
	verschluessselung<<<blocks, 1>>>(dev_kt_splitted, dev_gt_splitted);

	//zurueckkopieren
	//HANDLE_ERROR(cudaMemcpy(geheimtexte, dev_geheimtexte, sizeof(geheimtexte), cudaMemcpyDeviceToHost));
	
	//entschluesseln
	entschluessselung<<<blocks, 1>>>(dev_gt_splitted, dev_kt_splitted2);
	
	//zurueckkopieren
	HANDLE_ERROR(hipMemcpy(kt_splitted2, dev_kt_splitted2, sizeof(kt_splitted2), hipMemcpyDeviceToHost));
		
	//Ende der Zeitmessung
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	//Ausgabe der verstrichenen Zeit
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Elapsed time: %3.1f ms\n", elapsedTime);
	
	//ints wieder in char umwandeln
	unsplitt(klartext2, kt_splitted2);
	
	//Ausgabe
	printf("\n\nDer Klartext lautet nun:\n");
	for (i = 0; i < anzahl_Zeichen; i++)
	{
		putchar(klartext[i]);
	}
	printf("\n\n");

	//freigeben
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	return EXIT_SUCCESS;
}
