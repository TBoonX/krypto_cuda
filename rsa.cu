#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
//#include <math.h>
//https://mattmccutchen.net/bigint/
//#include "BigIntegerLibrary.hh"
//#define DEBUG

//Variablen
#define p 3
#define q 5
#define n 15
#define e 3
#define v 3
#define z 8
#define anzahl_Texte 80
#define count_cores 8

//__device__ long int klartexte[anzahl_Texte];
//__device__ long int klartexte_pruefung[anzahl_Texte];
//__device__ long int geheimtexte[anzahl_Texte];

/*
Klartext: K
Geheimtext: G
Verschluesselung: G = K^v mod n
Entschluesselung: K = G^e mod n

Index des CUDA Kerns: blockIdx.x blockIdx.y

*/

static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void verschluessselung(long int klartexte[], long int geheimtexte[])
{
	long int i, j, multi, x;
	
	long int block_length = anzahl_Texte/count_cores;
	
	for (i = 0 ; i < block_length; i++)
	{
		multi = x  = klartexte[i+blockIdx.x*block_length];
		for (j = 1; j < v; j++)
			x *= multi;
		
		geheimtexte[i+blockIdx.x*block_length] = x % n;
		
	}
}


__global__ void entschluessselung(long int geheimtexte[], long int klartexte_pruefung[])
{
	long int i, j, multi, x;
	
	long int block_length = anzahl_Texte/count_cores;

	for (i = 0 ; i < block_length; i++)
	{
		multi = x  = geheimtexte[i+blockIdx.x*block_length];
		for (j = 1; j < e; j++)
			x *= multi;
		
		klartexte_pruefung[i+blockIdx.x*block_length] = x % n;
		
	}
}

/*
int* equals(long int *a, long int *b)
{
	int i;

	for (i = 0; i < anzahl_Texte; i++)
	{
		if (a(i) != b(i))
			return 0;
	}
	return 1;
}
*/

int main(void) {
	int i;
	hipEvent_t start, stop;
	float elapsedTime;
	
	long int klartexte[anzahl_Texte];
	long int klartexte_pruefung[anzahl_Texte];
	long int geheimtexte[anzahl_Texte];
	
	printf("\nlong int: %d\n", sizeof(long int));
	printf("\nunsigned long int: %d\n", sizeof(unsigned long int));
	printf("\nunsigned long long: %d\n", sizeof(unsigned long long));
	

	//Klartetexte Array belegen
	//rand initialisieren
	srand((unsigned)time(NULL));
	for (i = 0; i < anzahl_Texte; i ++)
	{
		klartexte[i] = rand() % 15;		//Zahlen nicht  zu gross waehlen
	}
	
	printf("Die Klartexte:\n");
	for (i = 0; i < anzahl_Texte; i++)
	{
		printf("%ld, ", klartexte[i]);
	}
	printf("\n\n");

	long int *dev_klartexte, *dev_geheimtexte, *dev_klartexte_pruefung;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));



	HANDLE_ERROR(hipEventRecord(start, 0));

        HANDLE_ERROR(hipMalloc((void **)&dev_klartexte, sizeof(klartexte)));
        HANDLE_ERROR(hipMalloc((void **)&dev_geheimtexte, sizeof(geheimtexte)));
        HANDLE_ERROR(hipMalloc((void **)&dev_klartexte_pruefung, sizeof(klartexte_pruefung)));

        HANDLE_ERROR(hipMemcpy(dev_klartexte, klartexte, sizeof(klartexte), hipMemcpyHostToDevice));

	dim3 blocks(count_cores, 1);

	verschluessselung<<<blocks, 1>>>(dev_klartexte, dev_geheimtexte);

        HANDLE_ERROR(hipMemcpy(geheimtexte, dev_geheimtexte, sizeof(geheimtexte), hipMemcpyDeviceToHost));
		
	printf("Die Klartexte wurden verschluesselt.\n\nGeheimtexte:\n");
	for (i = 0; i < anzahl_Texte; i++)
	{
		printf("%ld, ", geheimtexte[i]);
	}
	printf("\n\n");
	
	entschluessselung<<<blocks, 1>>>(dev_geheimtexte, dev_klartexte_pruefung);
	
	HANDLE_ERROR(hipMemcpy(klartexte_pruefung, dev_klartexte_pruefung, sizeof(klartexte_pruefung), hipMemcpyDeviceToHost));
		

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	
	printf("Die Geheimtexte wurden entschluesselt.\n\nKlartexte:\n");
		for (i = 0; i < anzahl_Texte; i++)
		{
			printf("%ld, ", klartexte_pruefung[i]);
		}
		printf("\n\n");

/*	printf("\nDie Klartexte sind ");
	if (equals(klartexte, klartexte_pruefung))
		printf("NICHT ");
	printf("identisch.\n\n");
*/
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Elapsed time: %3.1f ms\n", elapsedTime);



	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	

	return EXIT_SUCCESS;
}
