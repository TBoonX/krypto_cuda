
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define output 1

//Variablen
#define p 5
#define q 7
#define n 35
#define e 5
#define v 5
#define z 24
#define anzahl_Zeichen 2688
#define count_cores 384

/*
Klartext: K
Geheimtext: G
Verschluesselung: G = K^v mod n
Entschluesselung: K = G^e mod n
Index des CUDA Kerns: blockIdx.x blockIdx.y
Ein groesserer Text soll ver- und entschluesselt werden.
Dieser wird jedoch wie folgt veraendert: nur kleine Buchstaben, keine Sonderzeichen außer . und ,
Dafuer werden die einzellnen chars in Integer umgewandelt (<=30).
Somit ist eine Verarbeitung moeglich.
*/

static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void verschluessselung(long int klartexte[], long int geheimtexte[])
{
	long int j, multi, x;
	
	//Fuer 384 Bloecke mit Threads
	long int threads = anzahl_Zeichen/count_cores;

	multi = x  = klartexte[threadIdx.x+blockIdx.x*threads];
	for (j = 1; j < v; j++)
		x *= multi;
	
	geheimtexte[threadIdx.x+blockIdx.x*threads] = x % n;
}

__global__ void entschluessselung(long int geheimtexte[], long int klartexte_pruefung[])
{
	long int j, multi, x;
	
	//Fuer 384 Bloecke mit Threads
	long int threads = anzahl_Zeichen/count_cores;
	
	multi = x  = geheimtexte[threadIdx.x+blockIdx.x*threads];
	for (j = 1; j < e; j++)
		x *= multi;
	
	klartexte_pruefung[threadIdx.x+blockIdx.x*threads] = x % n;
}

void splitt(char text[], long int numbers[])
{
	int i;
	
	//Splitte Klartext
	for (i = 0; i < anzahl_Zeichen; i++)
	{
		long int number = (int)text[i];
		
		//char in int beginnend mit 0
		//Sonderzeichen
		if (number == 44)		//,
			number = 27;
		else if (number == 46)		//.
			number = 28;
		else if (number == 10)		//\n
			number = 29;
		else if (number == 32)		//' '
			number = 30;
		else				//a-z
			number -= 97;
		
		numbers[i] = number;
	}
}

void unsplitt(char text[], long int numbers[])
{
	int i;
	
	//Splitte Klartext
	for (i = 0; i < anzahl_Zeichen; i++)
	{
		long int number = numbers[i];
		char t;
		
		//int in char
		//Sonderzeichen
		if (number == 27)		//,
			t = ',';
		else if (number == 28)		//.
			number = t = '.';
		else if (number == 29)		//\n
			t = '\n';
		else if (number == 30)		//' '
			t = ' ';
		else				//a-z
			t = (char)(number+97);
			
		text[i] = t;
	}
}

int main(int argc, char *argv[]) {
	int i;
	hipEvent_t start, stop;
	float elapsedTime;
	int count_Threads;
	long int multi = 0;
	
	char klartext[anzahl_Zeichen+1];
	char klartext2[anzahl_Zeichen+1];
	long int kt_splitted[anzahl_Zeichen+1];
	long int kt_splitted2[anzahl_Zeichen+1];
	long int *dev_kt_splitted, *dev_kt_splitted2, *dev_gt_splitted;
	int size = sizeof(long int)*(anzahl_Zeichen+1);
	
	printf("\n-|| RSA mit CUDA ||-\n\n\n");
	
	//Klartetext erzeugen
	klartext[anzahl_Zeichen] = klartext2[anzahl_Zeichen] = '\0';
	strcpy(klartext, "hat der alte hexenmeister?sich doch einmal wegbegeben.?und nun sollen seine geister?auch nach meinem willen leben.?seine wort und werke?merkt ich und den brauch,?und mit geistesstaerke?tu ich wunder auch.?walle. walle?manche strecke,?dass, zum zwecke,?wasser fliesse?und mit reichem, vollem schwalle?zu dem bade sich ergiesse.?und nun komm, du alter besen.?nimm die schlechten lumpenhuellen ?bist schon lange knecht gewesen:?nun erfuelle meinen willen.?auf zwei beinen stehe,?oben sei ein kopf,?eile nun und gehe?mit dem wassertopf.?walle. walle?manche strecke,?dass, zum zwecke,?wasser fliesse?und mit reichem, vollem schwalle?zu dem bade sich ergiesse.?seht, er laeuft zum ufer nieder,?wahrlich. ist schon an dem flusse,?und mit blitzesschnelle wieder?ist er hier mit raschem gusse.?schon zum zweiten male.?wie das becken schwillt.?wie sich jede schale?voll mit wasser fuellt.?stehe. stehe.?denn wir haben?deiner gaben?vollgemessen. ?ach, ich merk es. wehe. wehe.?hab ich doch das wort vergessen.?ach, das wort, worauf am ende?er das wird, was er gewesen.?ach, er laeuft und bringt behende.?waerst du doch der alte besen.?immer neue guesse?bringt er schnell herein,?ach. und hundert fluesse?stuerzen auf mich ein.?nein, nicht laenger?kann ichs lassen ?will ihn fassen.?das ist tuecke.?ach. nun wird mir immer baenger.?welche mine. welche blicke.?o du ausgeburt der hoelle.?soll das ganze haus ersaufen??seh ich ueber jede schwelle?doch schon wasserstroeme laufen.?ein verruchter besen,?der nicht hoeren will.?stock, der du gewesen,?steh doch wieder still.?willst am ende?gar nicht lassen??will dich fassen,?will dich halten?und das alte holz behende?mit dem scharfen beile spalten.?seht da kommt er schleppend wieder.?wie ich mich nur auf dich werfe,?gleich, o kobold, liegst du nieder ?krachend trifft die glatte schaerfe.?wahrlich, brav getroffen.?seht, er ist entzwei.?und nun kann ich hoffen,?und ich atme frei.?wehe. wehe.?beide teile?stehn in eile?schon als knechte?voellig fertig in die hoehe.?helft mir, ach. ihr hohen maechte.?und sie laufen. nass und naesser?wirds im saal und auf den stufen.?welch entsetzliches gewaesser.?herr und meister. hoer mich rufen.  ?ach, da kommt der meister.?herr, die not ist gross.?die ich rief, die geister?werd ich nun nicht los.?in die ecke,?besen, besen.?seids gewesen.?denn als geister?ruft euch nur zu diesem zwecke,?erst hervor der alte meister.?                                                                                                                                                                                                                                                                                                    ");
	printf("\n\nDer Klartext ist %d Zeichen lang.\n", sizeof(klartext)/sizeof(char)-1);
	
	if (argc < 2)
	{
		printf("\nParameter fuer Groesse der Klartexted fehlt!\n");
		exit(0);
	}

	//lese Anzahl der Zeichen
	multi = atoi(argv[1]);

	if (multi < 1)
	{
		printf("\nAnzahl der Zeichen ist eine ganze positive Zahl.\n");
		exit(0);
	}
	
	printf("\nmulti: %d\n", multi);
	
	//Ausgabe
	if (output)
	{
		printf("\n\nAnfang des  Klartextes:\n\n");
		for (i = 0; i < anzahl_Zeichen; i++)
		{
			if (klartext[i] == '?')
				klartext[i] = '\n';
			if (i < 546)
				putchar(klartext[i]);
		}
		printf("\n\n");
	}
	
	//klartext2 mit a füllen
	for (i = 0; i < anzahl_Zeichen; i++)
	{
		klartext2[i] = 'a';
	}
	
	printf("Der Klartext wird nun verschluesselt und anschliessend entschluesselt.\n");
	
	//Chars in ints aufsplitten
	splitt(klartext, kt_splitted);
	
	//Variablen der Zeitmessung erstellen
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	
	//Start Zeitmessung
	HANDLE_ERROR(hipEventRecord(start, 0));
	
	//allokieren
	HANDLE_ERROR(hipMalloc((void **)&dev_kt_splitted, size));
	HANDLE_ERROR(hipMalloc((void **)&dev_kt_splitted2, size));
	HANDLE_ERROR(hipMalloc((void **)&dev_gt_splitted, size));
	
	//kopieren
	HANDLE_ERROR(hipMemcpy(dev_kt_splitted, kt_splitted, size, hipMemcpyHostToDevice));
	
	//Anzahl Threads pro Block
	count_Threads = anzahl_Zeichen/count_cores;
	
	for (i = 0; i < multi; i++)
	{
		//verschluesseln
		verschluessselung<<<count_cores, count_Threads>>>(dev_kt_splitted, dev_gt_splitted);
		
		//sync
		HANDLE_ERROR(hipDeviceSynchronize());
		
		//entschluesseln
		entschluessselung<<<count_cores, count_Threads>>>(dev_gt_splitted, dev_kt_splitted2);
		
		//sync
		HANDLE_ERROR(hipDeviceSynchronize());
	}
	
	//zurueckkopieren
	HANDLE_ERROR(hipMemcpy(kt_splitted2, dev_kt_splitted2, size, hipMemcpyDeviceToHost));
		
	//Ende der Zeitmessung
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	
	//Ausgabe der verstrichenen Zeit
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("\nBeendet.\n\n\nverstrichene Zeit: %3.1f ms\n", elapsedTime);
	
	//ints wieder in char umwandeln
	unsplitt(klartext2, kt_splitted2);
	
	//Ausgabe
	if (output)
	{
		printf("\n\nDer Klartext lautet nun: (Anfang)\n\n");
		for (i = 0; i < 545; i++)
		{
			putchar(klartext2[i]);
		}
		printf("\n\n");
	}
	
	//freigeben
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	HANDLE_ERROR(hipFree(dev_kt_splitted));
	HANDLE_ERROR(hipFree(dev_kt_splitted2));
	HANDLE_ERROR(hipFree(dev_gt_splitted));
	return EXIT_SUCCESS;
}
