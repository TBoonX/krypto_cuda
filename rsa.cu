#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

//#define DEBUG

//Variablen
#define p 3
#define q 5
#define n 15
#define e 3
#define v 3
#define z 8
#define anzahl_Texte 1000

__device __ long int klartexte[anzahl_Texte];
__device __ long int klartexte_pruefung[anzahl_Texte];
__device __ long int geheimtexte[anzahl_Texte];

/*
Klartext: K
Geheimtext: G
Verschluesselung: G = K^v mod n
Entschluesselung: K = G^e mod n

Index des CUDA Kerns: blockIdx.x blockIdx.y

*/

#ifdef DEBUG
	#define WIDTH 3
#else
	#define WIDTH 256
#endif

static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__device__ int getArrayElement(int *m, int x, int y, int width) {
	return m[y * width + x];
}

__device__ void setArrayElement(int *m, int x, int y, int width, int value) {
	m[y * width + x] = value;
}


__global__ void matmul_simple(int *matM, int *matN, int *matP) {
	int sum, i;
	int m, n;

	sum = 0;

	for(i = 0; i < WIDTH; i++) {
		m = getArrayElement(matM, blockIdx.x, i, WIDTH);
		n = getArrayElement(matN, i, blockIdx.y, WIDTH);

		sum += m * n;
	}

	setArrayElement(matP, blockIdx.x, blockIdx.y, WIDTH, sum);
}

__global__ void verschluessselung(int klartext)
{
	int i;
	
	for (i = 0 ; i < 100; i ++)
	{
		//Integer hoch 103 ist zu hoch!
		geheimtexte[i+blockIdx.x*100] = pow(klartexte[i+blockIdx.x*100],v) % n;
	}
	
	printf("\nProzessor %d hat verschluesselt.\n", blockIdx.x);
}


__global__ void entschluessselung(int index)
{
	int i;
	
	for (i = 0 ; i < 100; i ++)
	{
		//Integer hoch 103 ist zu hoch!
		klartexte[i+blockIdx.x*100] = pow(geheimtexte[i+blockIdx.x*100],e) % n;
	}
	
	printf("\nProzessor %d hat entschluesselt.\n", blockIdx.x);	
}



int main(void) {
	int i, j;
	hipEvent_t start, stop;
	float elapsedTime;

	//Klartetexte Array belegen
	//rand initialisieren
	srand((unsigned)time(NULL));
	for (i = 0; i < anzahl_Texte; i ++)
	{
		klartexte[i] = rand() % 10;		//Zahlen nicht  zu gro� w�hlen
	}

	int *dev_matM, *dev_matN, *dev_matP;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	for(i = 0; i < WIDTH; i++) {
		for(j = 0; j < WIDTH; j++) {
#ifdef DEBUG
			matM[i + j * WIDTH] = i * WIDTH + j;
			matN[i + j * WIDTH] = i + j * WIDTH;
#else
			matM[i + j * WIDTH] = rand();
			matN[i + j * WIDTH] = rand();
#endif
		}
	}

	HANDLE_ERROR(hipEventRecord(start, 0));

        HANDLE_ERROR(hipMalloc((void **)&dev_matM, sizeof(matM)));
        HANDLE_ERROR(hipMalloc((void **)&dev_matN, sizeof(matN)));
        HANDLE_ERROR(hipMalloc((void **)&dev_matP, sizeof(matP)));

        HANDLE_ERROR(hipMemcpy(dev_matM, matM, sizeof(matM), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(dev_matN, matN, sizeof(matN), hipMemcpyHostToDevice));

	dim3 blocks(WIDTH, WIDTH);

	matmul_simple<<<blocks, 1>>>(dev_matM, dev_matN, dev_matP);

        HANDLE_ERROR(hipMemcpy(matP, dev_matP, sizeof(matP), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Elapsed time: %3.1f ms\n", elapsedTime);

#ifdef DEBUG
	printf("MatM:\n");
	for(i = 0; i < WIDTH; i++) {
		for(j = 0; j < WIDTH; j++) {
			printf("%4d ", matM[i + j * WIDTH]);
		}
		printf("\n");
	}

	printf("MatN:\n");
	for(i = 0; i < WIDTH; i++) {
		for(j = 0; j < WIDTH; j++) {
			printf("%4d ", matN[i + j * WIDTH]);
		}
		printf("\n");
	}

	printf("MatP:\n");
	for(i = 0; i < WIDTH; i++) {
		for(j = 0; j < WIDTH; j++) {
			printf("%4d ", matP[i + j * WIDTH]);
		}
		printf("\n");
	}
#endif

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	

	return EXIT_SUCCESS;
}
