#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
//#include <math.h>

//#define DEBUG

//Variablen
#define p 3
#define q 5
#define n 15
#define e 3
#define v 3
#define z 8
#define anzahl_Zeichen 2688
#define count_cores 384

/*
Klartext: K
Geheimtext: G
Verschluesselung: G = K^v mod n
Entschluesselung: K = G^e mod n

Index des CUDA Kerns: blockIdx.x blockIdx.y

Ein groesserer Text soll ver- und entschluesselt werden.
Dieser wird jedoch wie folgt veraendert: nur kleine Buchstaben, keine Sonderzeichen außer . und ,
Dafuer werden die einzellnen chars in Integer umgewandelt (<=28) und dies halbiert und auf 3 Werte verteilt.
Somit ist eine Verarbeitung moeglich.

*/

static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void verschluessselung(long int klartexte[], long int geheimtexte[])
{
	long int i, j, multi, x;
	
	long int block_length = anzahl_Texte/count_cores;
	
	for (i = 0 ; i < block_length; i++)
	{
		multi = x  = klartexte[i+blockIdx.x*block_length];
		for (j = 1; j < v; j++)
			x *= multi;
		
		geheimtexte[i+blockIdx.x*block_length] = x % n;
		
	}
}


__global__ void entschluessselung(long int geheimtexte[], long int klartexte_pruefung[])
{
	long int i, j, multi, x;
	
	long int block_length = anzahl_Texte/count_cores;

	for (i = 0 ; i < block_length; i++)
	{
		multi = x  = geheimtexte[i+blockIdx.x*block_length];
		for (j = 1; j < e; j++)
			x *= multi;
		
		klartexte_pruefung[i+blockIdx.x*block_length] = x % n;
		
	}
}

int main(void) {
	int i;
	hipEvent_t start, stop;
	float elapsedTime;
	
	char klartext[anzahlZeichen+1];
	char klartext2[anzahlZeichen+1];
	

	//Klartetext erzeugen
	klartext = "hat der alte hexenmeister
sich doch einmal wegbegeben.
und nun sollen seine geister
auch nach meinem willen leben.
seine wort und werke
merkt ich und den brauch,
und mit geistesstaerke
tu ich wunder auch.
walle. walle
manche strecke,
dass, zum zwecke,
wasser fliesse
und mit reichem, vollem schwalle
zu dem bade sich ergiesse.
und nun komm, du alter besen.
nimm die schlechten lumpenhuellen;
bist schon lange knecht gewesen:
nun erfuelle meinen willen.
auf zwei beinen stehe,
oben sei ein kopf,
eile nun und gehe
mit dem wassertopf.
walle. walle
manche strecke,
dass, zum zwecke,
wasser fliesse
und mit reichem, vollem schwalle
zu dem bade sich ergiesse.
seht, er laeuft zum ufer nieder,
wahrlich. ist schon an dem flusse,
und mit blitzesschnelle wieder
ist er hier mit raschem gusse.
schon zum zweiten male.
wie das becken schwillt.
wie sich jede schale
voll mit wasser fuellt.
stehe. stehe.
denn wir haben
deiner gaben
vollgemessen. 
ach, ich merk es. wehe. wehe.
hab ich doch das wort vergessen.
ach, das wort, worauf am ende
er das wird, was er gewesen.
ach, er laeuft und bringt behende.
waerst du doch der alte besen.
immer neue guesse
bringt er schnell herein,
ach. und hundert fluesse
stuerzen auf mich ein.
nein, nicht laenger
kann ichs lassen;
will ihn fassen.
das ist tuecke.
ach. nun wird mir immer baenger.
welche mine. welche blicke.
o du ausgeburt der hoelle.
soll das ganze haus ersaufen?
seh ich ueber jede schwelle
doch schon wasserstroeme laufen.
ein verruchter besen,
der nicht hoeren will.
stock, der du gewesen,
steh doch wieder still.
willst am ende
gar nicht lassen?
will dich fassen,
will dich halten
und das alte holz behende
mit dem scharfen beile spalten.
seht da kommt er schleppend wieder.
wie ich mich nur auf dich werfe,
gleich, o kobold, liegst du nieder;
krachend trifft die glatte schaerfe.
wahrlich, brav getroffen.
seht, er ist entzwei.
und nun kann ich hoffen,
und ich atme frei.
wehe. wehe.
beide teile
stehn in eile
schon als knechte
voellig fertig in die hoehe.
helft mir, ach. ihr hohen maechte.
und sie laufen. nass und naesser
wirds im saal und auf den stufen.
welch entsetzliches gewaesser.
herr und meister. hoer mich rufen.  
ach, da kommt der meister.
herr, die not ist gross.
die ich rief, die geister
werd ich nun nicht los.
in die ecke,
besen, besen.
seids gewesen.
denn als geister
ruft euch nur zu diesem zwecke,
erst hervor der alte meister.
                                                                                                                                                                                                  ";

	printf("\n\nDer Klartext ist %d Zeichen lang.\n", sizeof(klartext)/sizeof(char));

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));



	HANDLE_ERROR(hipEventRecord(start, 0));

        //HANDLE_ERROR(hipMalloc((void **)&dev_klartexte, sizeof(klartexte)));
        //HANDLE_ERROR(hipMalloc((void **)&dev_geheimtexte, sizeof(geheimtexte)));
        //HANDLE_ERROR(hipMalloc((void **)&dev_klartexte_pruefung, sizeof(klartexte_pruefung)));

        //HANDLE_ERROR(hipMemcpy(dev_klartexte, klartexte, sizeof(klartexte), hipMemcpyHostToDevice));

	dim3 blocks(count_cores, 1);

	//verschluessselung<<<blocks, 1>>>(dev_klartexte, dev_geheimtexte);

        //HANDLE_ERROR(hipMemcpy(geheimtexte, dev_geheimtexte, sizeof(geheimtexte), hipMemcpyDeviceToHost));
	
	//entschluessselung<<<blocks, 1>>>(dev_geheimtexte, dev_klartexte_pruefung);
	
	//HANDLE_ERROR(hipMemcpy(klartexte_pruefung, dev_klartexte_pruefung, sizeof(klartexte_pruefung), hipMemcpyDeviceToHost));
		

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Elapsed time: %3.1f ms\n", elapsedTime);



	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	

	return EXIT_SUCCESS;
}
