
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

//Variablen
#define p 5
#define q 7
#define n 35
#define e 5
#define v 5
#define z 24
#define anzahl_Zeichen 2688
#define count_cores 384

/*
Klartext: K
Geheimtext: G
Verschluesselung: G = K^v mod n
Entschluesselung: K = G^e mod n

Index des CUDA Kerns: blockIdx.x blockIdx.y

Ein groesserer Text soll ver- und entschluesselt werden.
Dieser wird jedoch wie folgt veraendert: nur kleine Buchstaben, keine Sonderzeichen au�er . und ,
Dafuer werden die einzellnen chars in Integer umgewandelt.
Somit ist eine Verarbeitung moeglich.

*/

static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void verschluessselung(long int klartexte[], long int geheimtexte[])
{
	long int i, j, multi, x;
	
	/*
	 * Schleife f�r Blockgroesse 384 mit je einem Thread
	
	long int block_length = anzahl_Zeichen/count_cores;
	
	for (i = 0 ; i < block_length; i++)
	{
		multi = x  = klartexte[i+blockIdx.x*count_cores];
		for (j = 1; j < v; j++)
			x *= multi;
		
		geheimtexte[i+blockIdx.x*count_cores] = x % n;
	}
	*/
	
	
	//Fuer 384 Bloecke mit 7 Threads
	long int threads = anzahl_Zeichen/count_cores;
	
	multi = x  = klartexte[threadIdx.x+blockIdx.x*threads];
	for (j = 1; j < v; j++)
		x *= multi;
	
	geheimtexte[threadIdx.x+blockIdx.x*threads] = x % n;
}


__global__ void entschluessselung(long int geheimtexte[], long int klartexte_pruefung[])
{
	long int i, j, multi, x;
	
	/*
	 * Schleife f�r Blockgroesse 384 mit je einem Thread
	
	long int block_length = anzahl_Zeichen/count_cores;

	for (i = 0 ; i < block_length; i++)
	{
		multi = x  = geheimtexte[i+blockIdx.x*count_cores];
		for (j = 1; j < e; j++)
			x *= multi;
		
		klartexte_pruefung[i+blockIdx.x*count_cores] = x % n;
	}
	*/
	
	//Fuer 384 Bloecke mit 7 Threads
	long int threads = anzahl_Zeichen/count_cores;
	
	multi = x  = geheimtexte[threadIdx.x+blockIdx.x*threads];
	for (j = 1; j < e; j++)
		x *= multi;
	
	klartexte_pruefung[threadIdx.x+blockIdx.x*threads] = x % n;
}

void splitt(char text[], long int numbers[])
{
	int i;
	
	//Splitte Klartext
	for (i = 0; i < anzahl_Zeichen; i++)
	{
		long int number = (int)text[i];
		
		//char in int beginnend mit 0
		//Sonderzeichen
		if (number == 44)		//,
			number = 27;
		else if (number == 46)		//.
			number = 28;
		else if (number == 10)		//\n
			number = 29;
		else if (number == 32)		//' '
			number = 30;
		else				//a-z
			number -= 97;
		
		numbers[i*3] = number;
	}
}

void unsplitt(char text[], long int numbers[])
{
	int i;
	
	//Splitte Klartext
	for (i = 0; i < anzahl_Zeichen; i++)
	{
		long int number = numbers[i*3];
		char t;
		
		//int in char
		//Sonderzeichen
		if (number == 27)		//,
			t = ',';
		else if (number == 28)		//.
			number = t = '.';
		else if (number == 29)		//\n
			t = '\n';
		else if (number == 30)		//' '
			t = ' ';
		else				//a-z
			t = (char)(number+97);
			
		text[i] = t;
	}
}

int main(void) {
	int i;
	hipEvent_t start, stop;
	float elapsedTime;
	int count_Threads;
	
	char klartext[anzahl_Zeichen+1];
	char klartext2[anzahl_Zeichen+1];
	long int kt_splitted[anzahl_Zeichen+1];
	long int kt_splitted2[anzahl_Zeichen+1];
	long int *dev_kt_splitted, *dev_kt_splitted2, *dev_gt_splitted;
	int size = sizeof(long int)*(anzahl_Zeichen+1);
	
	//Debug
	printf("\na: %d   z: %d   ,: %d   .: %d   ?: %d    : %d   backn: %d\n\n", (int)'a', (int)'z', (int)',', (int)'.', (int)'?', (int)' ',  (int)'\n');
	
	//TEST
	/*
	splitt(klartext, kt_splitted);
	for (i = 0; i < anzahl_Zeichen; i++)
	{
		klartext[i] = '-';
	}
	unsplitt(klartext, kt_splitted);
	*/

	//Klartetext erzeugen
	klartext[anzahl_Zeichen] = klartext2[anzahl_Zeichen] = '\0';
	strcpy(klartext, "hat der alte hexenmeister?sich doch einmal wegbegeben.?und nun sollen seine geister?auch nach meinem willen leben.?seine wort und werke?merkt ich und den brauch,?und mit geistesstaerke?tu ich wunder auch.?walle. walle?manche strecke,?dass, zum zwecke,?wasser fliesse?und mit reichem, vollem schwalle?zu dem bade sich ergiesse.?und nun komm, du alter besen.?nimm die schlechten lumpenhuellen ?bist schon lange knecht gewesen:?nun erfuelle meinen willen.?auf zwei beinen stehe,?oben sei ein kopf,?eile nun und gehe?mit dem wassertopf.?walle. walle?manche strecke,?dass, zum zwecke,?wasser fliesse?und mit reichem, vollem schwalle?zu dem bade sich ergiesse.?seht, er laeuft zum ufer nieder,?wahrlich. ist schon an dem flusse,?und mit blitzesschnelle wieder?ist er hier mit raschem gusse.?schon zum zweiten male.?wie das becken schwillt.?wie sich jede schale?voll mit wasser fuellt.?stehe. stehe.?denn wir haben?deiner gaben?vollgemessen. ?ach, ich merk es. wehe. wehe.?hab ich doch das wort vergessen.?ach, das wort, worauf am ende?er das wird, was er gewesen.?ach, er laeuft und bringt behende.?waerst du doch der alte besen.?immer neue guesse?bringt er schnell herein,?ach. und hundert fluesse?stuerzen auf mich ein.?nein, nicht laenger?kann ichs lassen ?will ihn fassen.?das ist tuecke.?ach. nun wird mir immer baenger.?welche mine. welche blicke.?o du ausgeburt der hoelle.?soll das ganze haus ersaufen??seh ich ueber jede schwelle?doch schon wasserstroeme laufen.?ein verruchter besen,?der nicht hoeren will.?stock, der du gewesen,?steh doch wieder still.?willst am ende?gar nicht lassen??will dich fassen,?will dich halten?und das alte holz behende?mit dem scharfen beile spalten.?seht da kommt er schleppend wieder.?wie ich mich nur auf dich werfe,?gleich, o kobold, liegst du nieder ?krachend trifft die glatte schaerfe.?wahrlich, brav getroffen.?seht, er ist entzwei.?und nun kann ich hoffen,?und ich atme frei.?wehe. wehe.?beide teile?stehn in eile?schon als knechte?voellig fertig in die hoehe.?helft mir, ach. ihr hohen maechte.?und sie laufen. nass und naesser?wirds im saal und auf den stufen.?welch entsetzliches gewaesser.?herr und meister. hoer mich rufen.  ?ach, da kommt der meister.?herr, die not ist gross.?die ich rief, die geister?werd ich nun nicht los.?in die ecke,?besen, besen.?seids gewesen.?denn als geister?ruft euch nur zu diesem zwecke,?erst hervor der alte meister.?                                                                                                                                                                                                                                                                                                    ");

	printf("\n\nDer Klartext ist %d Zeichen lang.\n", sizeof(klartext)/sizeof(char));
	
	//Ausgabe
	printf("\n\nDer Klartext:\n");
	for (i = 0; i < anzahl_Zeichen; i++)
	{
		if (klartext[i] == '?')
			klartext[i] = '\n';
		putchar(klartext[i]);
	}
	printf("\n\n");
	
	//klartext2 mit a f�llen
	for (i = 0; i < anzahl_Zeichen; i++)
	{
		klartext2[i] = 'a';
	}
	printf("Klartext2 mit a gefuellt\n");
	
	//Chars in ints aufsplitten
	splitt(klartext, kt_splitted);
	printf("klartext gesplitted\n");

	//Variablen der Zeitmessung erstellen
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	//Start Zeitmessung
	HANDLE_ERROR(hipEventRecord(start, 0));

	//allokieren
	HANDLE_ERROR(hipMalloc((void **)&dev_kt_splitted, sizeof(kt_splitted)));
	HANDLE_ERROR(hipMalloc((void **)&dev_kt_splitted2, sizeof(kt_splitted)));
	HANDLE_ERROR(hipMalloc((void **)&dev_gt_splitted, sizeof(kt_splitted)));
	printf("mit CUDA allokiert\n");

	//kopieren
	HANDLE_ERROR(hipMemcpy(dev_kt_splitted, kt_splitted, size, hipMemcpyHostToDevice));
	printf("mit CUDA kopiert\n");

	//Block festlegen
	//dim3 blocks(count_cores, 1);
	
	//Anzahl Threads pro Block
	count_Threads = anzahl_Zeichen/count_cores; 

	//verschluesseln
	verschluessselung<<<count_cores, count_Threads>>>(dev_kt_splitted, dev_gt_splitted);
	
	printf("\nVerschluesselung abgeschlossen ...\n\n");

	//zurueckkopieren
	//HANDLE_ERROR(cudaMemcpy(geheimtexte, dev_geheimtexte, sizeof(geheimtexte), cudaMemcpyDeviceToHost));
	
	//sync
	HANDLE_ERROR(hipDeviceSynchronize());
	
	//entschluesseln
	entschluessselung<<<count_cores, count_Threads>>>(dev_gt_splitted, dev_kt_splitted2);
	
	printf("\nEntschluesselung abgeschlossen ...\n\n");
	
	//zurueckkopieren
	HANDLE_ERROR(hipMemcpy(kt_splitted2, dev_kt_splitted2, size, hipMemcpyDeviceToHost));
		
	//Ende der Zeitmessung
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	//Ausgabe der verstrichenen Zeit
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Elapsed time: %3.1f ms\n", elapsedTime);
	
	//ints wieder in char umwandeln
	unsplitt(klartext2, kt_splitted2);
	
	//Ausgabe
	printf("\n\nDer Klartext lautet nun:\n");
	for (i = 0; i < anzahl_Zeichen; i++)
	{
		putchar(klartext2[i]);
	}
	printf("\n\n");

	//freigeben
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	HANDLE_ERROR(hipFree(dev_kt_splitted));
	HANDLE_ERROR(hipFree(dev_kt_splitted2));
	HANDLE_ERROR(hipFree(dev_gt_splitted));

	return EXIT_SUCCESS;
}

